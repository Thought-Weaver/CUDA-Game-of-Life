#include "hip/hip_runtime.h"
/**
 * Main CUDA file for running parallel cellular automaton.
 * @author Logan Apple
 * @date 5/15/2020
 */

#include "gol.cuh"

// What if I just passed the grid instead?
__host__ __device__ int count_neighbors(int x, int y, 
                                        int width, int height, 
                                        int* cells) {
    int alive = 0;
    
    if (x < 0 || x >= width || y < 0 || y >= height) {
        return 0;
    }

    for (int i = y - 1; i <= y + 1; ++i) {
        for (int j = x - 1; j <= x + 1; ++j) {
            if (i != y || j != x) {
                if (i >= 0 && i < height && j >= 0 && j < width) {
                    alive += cells[i * width + j];
                }
            }
        }
    }

    return alive;
}

__global__ void naive_update_kernel(int width, int height, 
                                    int* cells, int* updated_cells) {
    const int num_threads_x = blockDim.x * gridDim.x;
    const int num_threads_y = blockDim.y * gridDim.y;

    // Thread indices.
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (; tidy < height; tidy += num_threads_y) {
        for (; tidx < width; tidx += num_threads_x) {
            int neighbors = count_neighbors(tidx, tidy, width, height, cells);
            // Any live cell with two or three neighbors survives.
            if (cells[tidy * width + tidx] == 1 && 
                (neighbors == 2 || neighbors == 3)) {
                updated_cells[tidy * width + tidx] = 1;
            }
            // Any dead cell with three live neighbors comes to life.
            else if (cells[tidy * width + tidx] == 0 && neighbors == 3) {
                updated_cells[tidy * width + tidx] = 1;
            }
            // Any other cells die.
            else {
                updated_cells[tidy * width + tidx] = 0;
            }
        }
    }
}

__global__ void optimized_update_kernel(int width, int height, 
                                        int* cells, int* updated_cells) {

}

void call_cuda_gol_update(int blocks,
                          int width, int height,
                          int* cells, int* updated_cells,
                          bool optimized) {
    // Maybe I should fix these rather than let the user specify them?
    dim3 block_size(blocks, blocks);
    dim3 grid_size(int((width + blocks - 1) / blocks), 
                   int((height + blocks - 1) / blocks));
    if (optimized) {
        optimized_update_kernel<<<grid_size, block_size>>>(width, height, 
                                                        cells, updated_cells);
    }
    else {
        naive_update_kernel<<<grid_size, block_size>>>(width, height,
                                                    cells, updated_cells);
    }
}