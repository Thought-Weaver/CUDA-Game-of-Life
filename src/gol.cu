/**
 * Main CUDA file for running parallel cellular automaton.
 * @author Logan Apple
 * @date 5/15/2020
 */

#include "gol.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>

