#include "hip/hip_runtime.h"
/**
 * Main CUDA file for running parallel cellular automaton.
 * @author Logan Apple
 * @date 5/15/2020
 */

#include "gol.cuh"

// Trying texture memory -- probably not better than just using shared memory,
// but it's worth a shot.
// Also, I feel like this shouldn't be global, but all the CUDA examples have
// it as such?
texture<uint8_t, 2, hipReadModeElementType> texmem;

// What if I just passed the grid instead?
__host__ __device__ uint8_t count_neighbors(int x, int y, 
                                        int width, int height, 
                                        uint8_t* cells) {
    uint8_t alive = 0;
    
    if (x < 0 || x >= width || y < 0 || y >= height) {
        return 0;
    }

    for (int i = y - 1; i <= y + 1; ++i) {
        for (int j = x - 1; j <= x + 1; ++j) {
            if (i != y || j != x) {
                if (i >= 0 && i < height && j >= 0 && j < width) {
                    alive += cells[i * width + j];
                }
            }
        }
    }

    return alive;
}

__global__ void naive_update_kernel(int width, int height, 
                                    uint8_t* cells, uint8_t* updated_cells) {
    const int num_threads_x = blockDim.x * gridDim.x;
    const int num_threads_y = blockDim.y * gridDim.y;

    // Thread indices.
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (; tidy < height; tidy += num_threads_y) {
        for (; tidx < width; tidx += num_threads_x) {
            uint8_t neighbors = count_neighbors(tidx, tidy, width, height, cells);
            // Any live cell with two or three neighbors survives.
            if (cells[tidy * width + tidx] == 1 && 
                    (neighbors == 2 || neighbors == 3)) {
                updated_cells[tidy * width + tidx] = 1;
            }
            // Any dead cell with three live neighbors comes to life.
            else if (cells[tidy * width + tidx] == 0 && neighbors == 3) {
                updated_cells[tidy * width + tidx] = 1;
            }
            // Any other cells die.
            else {
                updated_cells[tidy * width + tidx] = 0;
            }
        }
    }
}

__global__ void optimized_update_kernel(int width, int height,
                                        uint8_t* updated_cells) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    uint8_t neighbors = 0;
            
    // Take advantage of loop unrolling to make this faster.
    #pragma unroll
    for (int x = -1; x <= 1; ++x) {
        #pragma unroll
        for (int y = -1; y <= 1; ++y) {
            int y2 = tidy + y;
            int x2 = tidx + x;
            if (y2 != y || x2 != x) {
                if (y2 >= 0 && y2 < height && 
                    x2 >= 0 && x2 < width) {
                    neighbors += tex2D(texmem, x2, y2);
                }
            }
        }
    }

    // Any live cell with two or three neighbors survives.
    if (tex2D(texmem, tidx, tidy) == 1 && 
            (neighbors == 2 || neighbors == 3)) {
        updated_cells[tidy * width + tidx] = 1;
    }
    // Any dead cell with three live neighbors comes to life.
    else if (tex2D(texmem, tidx, tidy) == 0 && neighbors == 3) {
        updated_cells[tidy * width + tidx] = 1;
    }
    // Any other cells die.
    else {
        updated_cells[tidy * width + tidx] = 0;
    }
}

void call_cuda_gol_update(int num_threads,
                          int width, int height,
                          uint8_t* cells, uint8_t* updated_cells,
                          bool optimized) {
    // Maybe I should fix these rather than let the user specify them?
    dim3 block_size(num_threads, num_threads);
    dim3 grid_size(int((width + num_threads - 1) / num_threads), 
                   int((height + num_threads - 1) / num_threads));
    if (optimized) {
        hipChannelFormatDesc desc = hipCreateChannelDesc<uint8_t>();

        size_t pitch;
        hipMallocPitch(&cells, &pitch, sizeof(uint8_t) * width, height);
        hipBindTexture2D(0, texmem, cells, desc, width, height, pitch);

        optimized_update_kernel<<<grid_size, block_size>>>(width, height, 
            updated_cells);
    }
    else {
        naive_update_kernel<<<grid_size, block_size>>>(width, height, 
            cells, updated_cells);
    }
}