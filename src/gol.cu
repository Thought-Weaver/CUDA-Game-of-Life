#include "hip/hip_runtime.h"
/**
 * Main CUDA file for running parallel cellular automaton.
 * @author Logan Apple
 * @date 5/15/2020
 */

#include "gol.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void naive_gol_update_kernel();

__global__ void cuda_optimized_gol_update_kernel();

void call_cuda_gol_update();