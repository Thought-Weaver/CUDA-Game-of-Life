#include "hip/hip_runtime.h"
/**
 * Main CUDA file for running parallel cellular automaton.
 * @author Logan Apple
 * @date 5/15/2020
 */

#include "gol.cuh"

// What if I just passed the grid instead?
__host__ __device__ uint8_t count_neighbors(int x, int y, 
                                        int width, int height, 
                                        uint8_t* cells) {
    uint8_t alive = 0;
    
    if (x < 0 || x >= width || y < 0 || y >= height) {
        return 0;
    }

    for (int i = y - 1; i <= y + 1; ++i) {
        for (int j = x - 1; j <= x + 1; ++j) {
            if (i != y || j != x) {
                if (i >= 0 && i < height && j >= 0 && j < width) {
                    alive += cells[i * width + j];
                }
            }
        }
    }

    return alive;
}

__global__ void naive_update_kernel(int width, int height, 
                                    uint8_t* cells, uint8_t* updated_cells) {
    const int num_threads_x = blockDim.x * gridDim.x;
    const int num_threads_y = blockDim.y * gridDim.y;

    // Thread indices.
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (; tidy < height; tidy += num_threads_y) {
        for (; tidx < width; tidx += num_threads_x) {
            uint8_t neighbors = count_neighbors(tidx, tidy, width, height, cells);
            // Any live cell with two or three neighbors survives.
            if (cells[tidy * width + tidx] == 1 && 
                    (neighbors == 2 || neighbors == 3)) {
                updated_cells[tidy * width + tidx] = 1;
            }
            // Any dead cell with three live neighbors comes to life.
            else if (cells[tidy * width + tidx] == 0 && neighbors == 3) {
                updated_cells[tidy * width + tidx] = 1;
            }
            // Any other cells die.
            else {
                updated_cells[tidy * width + tidx] = 0;
            }
        }
    }
}

__global__ void optimized_update_kernel(int width, int height,
        uint8_t* cells, uint8_t* updated_cells) {
    extern __shared__ uint8_t shmem[];

    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    
    int i = threadIdx.y;
    int j = threadIdx.x;

    if (tidx >= 0 && tidx < width && tidy >= 0 && tidy < height) {
        shmem[i * width + j] = cells[tidy * width + tidx];
    }

    __syncthreads();

    if (tidx >= 0 && tidx < width && tidy >= 0 && tidy < height) {
        uint8_t neighbors = 0;

        // Take advantage of loop unrolling to make this faster.
        #pragma unroll
        for (int x = -1; x <= 1; ++x) {
            #pragma unroll
            for (int y = -1; y <= 1; ++y) {
                int y2 = i + y;
                int x2 = j + x;
                if (x != 0 || y != 0) {
                    if (y2 >= 0 && y2 < height && 
                            x2 >= 0 && x2 < width) {
                        neighbors += shmem[y2 * width + x2];
                    }
                }
            }
        }

        // Any live cell with two or three neighbors survives.
        if ((neighbors == 2 || neighbors == 3) && shmem[i * width + j] == 1) {
            updated_cells[tidy * width + tidx] = 1;
        }
        // Any dead cell with three live neighbors comes to life.
        else if (neighbors == 3 && shmem[i * width + j] == 0) {
            updated_cells[tidy * width + tidx] = 1;
        }
        // Any other cells die.
        else {
            updated_cells[tidy * width + tidx] = 0;
        }
    }
}

__global__ void optimized_update_kernel_bitwise(int width, int height, 
                                    uint8_t* cells, uint8_t* updated_cells) {
    const int num_threads_x = blockDim.x * gridDim.x;
    const int num_threads_y = blockDim.y * gridDim.y;

    // Thread indices.
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (; tidy < height; tidy += num_threads_y) {
        for (; tidx < width; tidx += num_threads_x) {
            for (int k = 0; k < 8; ++k) {
                uint8_t current = (cells[tidy * width + tidx] & (1 << k)) >> k;
                uint8_t top_left = 0,
                        top_mid = 0,
                        top_right = 0,
                        mid_left = 0,
                        mid_right = 0,
                        bot_left = 0,
                        bot_mid = 0,
                        bot_right = 0;
                
                // If there's a top-left relative to the current position.
                if (tidy > 0) {
                    // If k is 0, then we need the previous set of 8 cells, else
                    // we can just use the previous bit in the current set.
                    if (k == 0 && tidx > 0) {
                        top_left = (cells[(tidy - 1) * width + (tidx - 1)] & 
                            (1 << 7)) >> 7;
                    }
                    else {
                        top_left = (cells[(tidy - 1) * width + tidx] & 
                            (1 << (k - 1))) >> (k - 1);
                    }
                }
    
                // If there's a top relative to the current position.
                if (tidy > 0) {
                    top_mid = (cells[(tidy - 1) * width + tidx] & 
                        (1 << k)) >> k;
                }
    
                // If there's a top-right relative to the current position.
                if (tidy > 0) {
                    // If k is 7, then we need the next set of 8 cells, else
                    // we can just use the next bit in the current set.
                    if (tidx < width - 1 && k == 7) {
                        top_right = (cells[(tidy - 1) * width + (tidx + 1)] & 
                            (1 << 0)) >> 0;
                    }
                    else {
                        top_right = (cells[(tidy - 1) * width + tidx] & 
                            (1 << (k + 1))) >> (k + 1);
                    }
                }
    
                // If there's a left relative to the current position.
                if (tidx > 0 && k == 0) {
                    // If k is 0, then we need the previous set of 8 cells, else
                    // we can just use the previous bit in the current set.
                    mid_left = (cells[tidy * width + (tidx - 1)] & 
                        (1 << 7)) >> 7;
                }
                else {
                    mid_left = (cells[tidy * width + tidx] & 
                        (1 << (k - 1))) >> (k - 1);
                }
    
                // If there's a right relative to the current position.
                if (k == 7 && tidx < width - 1) {
                    // If k is 7, then we need the next set of 8 cells, else
                    // we can just use the next bit in the current set.
                    mid_right = (cells[tidy * width + (tidx + 1)] 
                        & (1 << 0)) >> 0;
                }
                else {
                    mid_right = (cells[tidy * width + tidx] & 
                        (1 << (k + 1))) >> (k + 1);
                }
    
                // If there's a bottom-left relative to the current position.
                if (tidy < height - 1) {
                    // If k is 0, then we need the previous set of 8 cells, else
                    // we can just use the previous bit in the current set.
                    if (k == 0 && tidx > 0) {
                        bot_left = (cells[(tidy + 1) * width + (tidx - 1)] & 
                            (1 << 7)) >> 7;
                    }
                    else {
                        bot_left = (cells[(tidy + 1) * width + tidx] & 
                            (1 << (k - 1))) >> (k - 1);
                    }
                }
    
                // If there's a bottom relative to the current position.
                if (tidy < height - 1) {
                    bot_mid = (cells[(tidy + 1) * width + tidx] & 
                        (1 << k)) >> k;
                }
    
                // If there's a bottom-right relative to the current position.
                if (tidy < height - 1) {
                    // If k is 7, then we need the next set of 8 cells, else
                    // we can just use the next bit in the current set.
                    if (k == 7 && tidx < width - 1) {
                        bot_right = (cells[(tidy + 1) * width + (tidx + 1)] & 
                            (1 << 0)) >> 0;
                    }
                    else {
                        bot_right = (cells[(tidy + 1) * width + tidx] & 
                            (1 << (k + 1))) >> (k + 1);
                    }
                }
    
                uint8_t neighbors = top_left + top_mid + top_right + 
                                    mid_left +           mid_right + 
                                    bot_left + bot_mid + bot_right;

                // Any live cell with two or three neighbors survives.
                if ((neighbors == 2 || neighbors == 3) && current == 1) {
                    updated_cells[tidy * width + tidx] |= 1 << k;
                }
                // Any dead cell with three live neighbors comes to life.
                else if (neighbors == 3 && current == 0) {
                    updated_cells[tidy * width + tidx] |= 1 << k;
                }
            }
        }
    }
}

void call_cuda_gol_update(int num_threads,
                          int width, int height,
                          uint8_t* cells, uint8_t* updated_cells,
                          bool optimized) {
    int actual_width = width % 8 == 0 ? width / 8 : width;
    int x_blocks = (actual_width + num_threads - 1) / num_threads;
    int y_blocks = (height + num_threads - 1) / num_threads;

    dim3 block_size(num_threads, num_threads);
    dim3 grid_size(x_blocks, y_blocks);

    if (optimized) {
        if (width % 8 == 0) {
            optimized_update_kernel_bitwise<<<grid_size, block_size>>>
                (actual_width, height, cells, updated_cells);
        }
        else {
            optimized_update_kernel<<<grid_size, block_size, 
                (num_threads + 2) * (num_threads + 2) * sizeof(uint8_t)>>>
                (width, height, cells, updated_cells);
        }
    }
    else {
        naive_update_kernel<<<grid_size, block_size>>>(width, height, 
            cells, updated_cells);
    }
}